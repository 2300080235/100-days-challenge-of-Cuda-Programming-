#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int idx = row * N + col;
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    const int N = 10;
    const int size = N * N * sizeof(float);

    // Allocate host memory
    float *A = (float*)malloc(size);
    float *B = (float*)malloc(size);
    float *C = (float*)malloc(size);

    // Initialize input matrices
    for (int i = 0; i < N * N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
        C[i] = 0.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define execution configuration
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    // Launch kernel
    matrixAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Matrix C (Result):" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Clean up
    free(A); free(B); free(C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
